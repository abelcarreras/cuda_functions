#include "hip/hip_runtime.h"
#include <Python.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <numpy/arrayobject.h>
#include <string.h>
#include <math.h>
#include <complex.h>


// includes, project
#include <hipblas.h>

// Complex data type
typedef float2 Complex;
typedef double2 DoubleComplex;


///////////////////////////////////////////////////////////
//                                                       //
// Single precision autocorrelation (Complex to complex) //
//                                                       //
///////////////////////////////////////////////////////////

static PyObject* autocorrelation_sp(PyObject* self, PyObject *arg, PyObject *keywords)
{
    const char *Mode = "single";   // Default value of Mode (to mimic numpy behavior)

    //  Interface with Python
    PyObject *h_signal_obj;

    static char *kwlist[] = {"input_data", "mode", NULL};
    if (!PyArg_ParseTupleAndKeywords(arg, keywords, "O|s", kwlist, &h_signal_obj, &Mode))  return NULL;

    PyObject *h_signal_array = PyArray_FROM_OTF(h_signal_obj, NPY_CFLOAT, NPY_IN_ARRAY);

    if (h_signal_array == NULL ) {
         Py_XDECREF(h_signal_array);
         return NULL;
    }

    Complex *h_signal = (Complex *)PyArray_DATA(h_signal_array);
    int     SignalSize = (int)PyArray_DIM(h_signal_array, 0);

    // Output intermediate variable
    Complex h_output;

    // Allocate device memory for signal
    Complex* d_signal;
    hipMalloc((void**)&d_signal, sizeof(Complex) * SignalSize);

    // Copy host memory to device
    hipMemcpy(d_signal, h_signal, sizeof(Complex) * SignalSize, hipMemcpyHostToDevice);

    // Create a handle for CUBLAS
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Prepare output python object
    PyArrayObject *return_object;

    if  (strcmp(Mode, "full") == 0) {

        // Prepare output numpy array
        int dims[1]={SignalSize*2-1};
        return_object = (PyArrayObject *) PyArray_FromDims(1,dims,NPY_CFLOAT);
        Complex *Return_data  = (Complex *)PyArray_DATA(return_object);

        // Dot product using cuBlas
        for (int i=0; i< SignalSize; i++){
            // Dot product using cuBlas
            hipblasCdotc(handle, SignalSize-i,
                         &d_signal[i], 1,
                         d_signal, 1,
                         &h_output);
            Return_data[(SignalSize*2-1)/2-i] = h_output;
            if (((SignalSize*2-1)/2+i) < SignalSize*2-1) Return_data[(SignalSize*2-1)/2+i] = h_output;
        }
    }
    else if  (strcmp(Mode, "same") == 0) {

        // Prepare output numpy array
        int dims[1]={SignalSize};
        return_object = (PyArrayObject *) PyArray_FromDims(1,dims,NPY_CFLOAT);
        Complex *Return_data  = (Complex *)PyArray_DATA(return_object);

        for (int i=0; i< SignalSize/2+1; i++){
            // Dot product using cuBlas
            hipblasCdotc(handle, SignalSize-i,
                         &d_signal[i], 1,
                         d_signal, 1,
                         &h_output);
            Return_data[SignalSize/2-i] = h_output;
            if ((SignalSize/2+i) < SignalSize) Return_data[SignalSize/2+i] = h_output;
        }
    }
    else if  (strcmp(Mode, "single") == 0) {

        // Prepare output numpy array
        int dims[1]={1};
        return_object = (PyArrayObject *) PyArray_FromDims(1,dims,NPY_CFLOAT);
        Complex *Return_data  = (Complex *)PyArray_DATA(return_object);

        // Dot product using cuBlas
        hipblasCdotc(handle, SignalSize,
                     d_signal, 1,
                     d_signal, 1,
                     &h_output);
        Return_data[0] = h_output;
    }
   else {
        PyErr_SetString(PyExc_TypeError, "this mode do not exist");
        PyErr_Print();
    }

    // cleanup memory device
    hipFree(d_signal);

    // Finish cublas
    hipblasDestroy(handle);

    // Clean up memory python
    Py_DECREF(h_signal_array);

    //Returning Python array
    return(PyArray_Return(return_object));
}


///////////////////////////////////////////////////////////
//                                                       //
// Double precision autocorrelation (Complex to complex) //
//                                                       //
///////////////////////////////////////////////////////////

static PyObject* autocorrelation_dp(PyObject* self, PyObject *arg, PyObject *keywords)
{
    const char    *Mode = "single";   // Default value of Mode (to mimic numpy behavior)

    //  Interface with Python
    PyObject *h_signal_obj;

    static char *kwlist[] = {"input_data", "mode", NULL};
    if (!PyArg_ParseTupleAndKeywords(arg, keywords, "O|s", kwlist, &h_signal_obj, &Mode))  return NULL;

    PyObject *h_signal_array = PyArray_FROM_OTF(h_signal_obj, NPY_CDOUBLE, NPY_IN_ARRAY);

    if (h_signal_array == NULL ) {
         Py_XDECREF(h_signal_array);
         return NULL;
    }

    DoubleComplex *h_signal = (DoubleComplex *)PyArray_DATA(h_signal_array);
    int     SignalSize = (int)PyArray_DIM(h_signal_array, 0);

    // Output intermediate variable
    DoubleComplex h_output;

    // Allocate device memory for signal
    DoubleComplex* d_signal;
    hipMalloc((void**)&d_signal, sizeof(DoubleComplex) * SignalSize);

    // Copy host memory to device
    hipMemcpy(d_signal, h_signal, sizeof(DoubleComplex) * SignalSize, hipMemcpyHostToDevice);

    // Create a handle for CUBLAS
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Prepare output python object
    PyArrayObject *return_object;

    if  (strcmp(Mode, "full") == 0) {

        // Prepare output numpy array
        int dims[1]={SignalSize*2-1};
        return_object = (PyArrayObject *) PyArray_FromDims(1,dims,NPY_CDOUBLE);
        DoubleComplex *Return_data  = (DoubleComplex *)PyArray_DATA(return_object);

        // Dot product using cuBlas
        for (int i=0; i< SignalSize; i++){
            // Dot product using cuBlas
            hipblasZdotc(handle, SignalSize-i,
                         &d_signal[i], 1,
                         d_signal, 1,
                         &h_output);
            if (((SignalSize*2-1)/2+i) < SignalSize*2-1) Return_data[(SignalSize*2-1)/2+i] = h_output;
            Return_data[(SignalSize*2-1)/2-i] = h_output;
        }
    }
    else if  (strcmp(Mode, "same") == 0) {

        // Prepare output numpy array
        int dims[1]={SignalSize};
        return_object = (PyArrayObject *) PyArray_FromDims(1,dims,NPY_CDOUBLE);
        DoubleComplex *Return_data  = (DoubleComplex *)PyArray_DATA(return_object);

        for (int i=0; i< SignalSize/2+1; i++){

            // Dot product using cuBlas
            hipblasZdotc(handle, SignalSize-i,
                         &d_signal[i], 1,
                         d_signal, 1,
                         &h_output);
             //   printf("%d: %lf\n ", SignalSize/2+i, h_output);

            if ((SignalSize/2+i) < SignalSize) Return_data[SignalSize/2+i] = h_output;
            Return_data[SignalSize/2-i] = h_output;
        }

    }
    else if  (strcmp(Mode, "single") == 0) {

        // Prepare output numpy array
        int dims[1]={1};
        return_object = (PyArrayObject *) PyArray_FromDims(1,dims,NPY_CDOUBLE);
        DoubleComplex *Return_data  = (DoubleComplex *)PyArray_DATA(return_object);

        // Dot product using cuBlas
        hipblasZdotc(handle, SignalSize,
                     d_signal, 1,
                     d_signal, 1,
                     &h_output);
        Return_data[0] = h_output;
    }
   else {
        PyErr_SetString(PyExc_TypeError, "this mode do not exist");
        PyErr_Print();
   }

    // Finish cublas
    hipblasDestroy(handle);

    // cleanup memory device
    hipFree(d_signal);

    // Clean up memory python
    Py_DECREF(h_signal_array);

    //Returning Python array
    return(PyArray_Return(return_object));
//    return(h_signal_array);
}



static char extension_docs_sp[] =
    "autocorrelation(signal)\nAutocorrelation implemented in CUDA\n(Single precision)\n  ";

static char extension_docs_dp[] =
    "autocorrelation(signal)\nAutocorrelation implemented in CUDA\n(Double precision)\n  ";


static PyMethodDef extension_funcs[] =
{
    {"acorrelate", (PyCFunction) autocorrelation_sp, METH_VARARGS|METH_KEYWORDS, extension_docs_sp},
    {"dacorrelate", (PyCFunction) autocorrelation_dp, METH_VARARGS|METH_KEYWORDS, extension_docs_dp},
    {NULL}
};


PyMODINIT_FUNC initgpu_correlate(void)
{
//  Importing numpy array types
    import_array();
    Py_InitModule3("gpu_correlate", extension_funcs,
                   "Autocorrelation functions (CUDA)");
};