#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include <complex.h>

#include <Python.h>
#include <numpy/arrayobject.h>

#include <hipblas.h>

// Complex data type
typedef float2 Complex;
typedef double2 DoubleComplex;


static PyObject* autocorrelation_sp(PyObject* self, PyObject *arg, PyObject *keywords);
static PyObject* autocorrelation_dp(PyObject* self, PyObject *arg, PyObject *keywords);


///////////////////////////////////////////////////////////
//                                                       //
// Single precision autocorrelation (Complex to complex) //
//                                                       //
///////////////////////////////////////////////////////////

static PyObject* autocorrelation_sp(PyObject* self, PyObject *arg, PyObject *keywords)
{
    const char *mode = "valid";   // Default value of mode (to mimic numpy behavior)

    //  Interface with Python
    PyObject *h_signal_obj;

    static char *kwlist[] = {"input_data", "mode", NULL};
    if (!PyArg_ParseTupleAndKeywords(arg, keywords, "O|s", kwlist, &h_signal_obj, &mode))  return NULL;

    PyObject *h_signal_array = PyArray_FROM_OTF(h_signal_obj, NPY_CFLOAT, NPY_IN_ARRAY);

    if (h_signal_array == NULL ) {
         Py_XDECREF(h_signal_array);
         return NULL;
    }

    Complex *h_signal = (Complex *)PyArray_DATA(h_signal_array);
    int     signal_size = (int)PyArray_DIM(h_signal_array, 0);

    Complex h_output;

    // Allocate device memory for signal
    Complex* d_signal;
    hipMalloc((void**)&d_signal, sizeof(Complex) * signal_size);

    // Copy host memory to device
    hipMemcpy(d_signal, h_signal, sizeof(Complex) * signal_size, hipMemcpyHostToDevice);

    // Create a handle for CUBLAS
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Prepare output python object
    PyArrayObject *return_object;

    if  (strcmp(mode, "full") == 0) {

        int dims[1]={signal_size*2-1};
        return_object = (PyArrayObject *) PyArray_FromDims(1,dims,NPY_CFLOAT);
        Complex *return_data  = (Complex *)PyArray_DATA(return_object);

        for (int i=0; i< signal_size; i++){
            hipblasCdotc(handle, signal_size-i,
                         &d_signal[i], 1,
                         d_signal, 1,
                         &h_output);
            return_data[(signal_size*2-1)/2-i] = h_output;
            if (((signal_size*2-1)/2+i) < signal_size*2-1) return_data[(signal_size*2-1)/2+i] = h_output;
        }
    }
    else if  (strcmp(mode, "same") == 0) {

        int dims[1]={signal_size};
        return_object = (PyArrayObject *) PyArray_FromDims(1,dims,NPY_CFLOAT);
        Complex *return_data  = (Complex *)PyArray_DATA(return_object);

        for (int i=0; i< signal_size/2+1; i++){
            hipblasCdotc(handle, signal_size-i,
                         &d_signal[i], 1,
                         d_signal, 1,
                         &h_output);
            return_data[signal_size/2-i] = h_output;
            if ((signal_size/2+i) < signal_size) return_data[signal_size/2+i] = h_output;
        }
    }
    else if  (strcmp(mode, "valid") == 0) {

        int dims[1]={1};
        return_object = (PyArrayObject *) PyArray_FromDims(1,dims,NPY_CFLOAT);
        Complex *return_data  = (Complex *)PyArray_DATA(return_object);

        hipblasCdotc(handle, signal_size,
                     d_signal, 1,
                     d_signal, 1,
                     &h_output);
        return_data[0] = h_output;
    }
   else {
        PyErr_SetString(PyExc_TypeError, "this mode do not exist");
        PyErr_Print();
    }

    // cleanup memory
    hipFree(d_signal);
    hipblasDestroy(handle);
    Py_DECREF(h_signal_array);

    return(PyArray_Return(return_object));
}


///////////////////////////////////////////////////////////
//                                                       //
// Double precision autocorrelation (Complex to complex) //
//                                                       //
///////////////////////////////////////////////////////////

static PyObject* autocorrelation_dp(PyObject* self, PyObject *arg, PyObject *keywords)
{
    const char    *mode = "valid";   // Default value of mode (to mimic numpy behavior)

    //  Interface with Python
    PyObject *h_signal_obj;

    static char *kwlist[] = {"input_data", "mode", NULL};
    if (!PyArg_ParseTupleAndKeywords(arg, keywords, "O|s", kwlist, &h_signal_obj, &mode))  return NULL;

    PyObject *h_signal_array = PyArray_FROM_OTF(h_signal_obj, NPY_CDOUBLE, NPY_IN_ARRAY);

    if (h_signal_array == NULL ) {
         Py_XDECREF(h_signal_array);
         return NULL;
    }

    DoubleComplex *h_signal = (DoubleComplex *)PyArray_DATA(h_signal_array);
    int     signal_size = (int)PyArray_DIM(h_signal_array, 0);

    DoubleComplex h_output;

    // Allocate device memory for signal
    DoubleComplex* d_signal;
    hipMalloc((void**)&d_signal, sizeof(DoubleComplex) * signal_size);

    // Copy host memory to device
    hipMemcpy(d_signal, h_signal, sizeof(DoubleComplex) * signal_size, hipMemcpyHostToDevice);

    // Create a handle for CUBLAS
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Prepare output python object
    PyArrayObject *return_object;

    if  (strcmp(mode, "full") == 0) {

        int dims[1]={signal_size*2-1};
        return_object = (PyArrayObject *) PyArray_FromDims(1,dims,NPY_CDOUBLE);
        DoubleComplex *return_data  = (DoubleComplex *)PyArray_DATA(return_object);

        for (int i=0; i< signal_size; i++){
            hipblasZdotc(handle, signal_size-i,
                         &d_signal[i], 1,
                         d_signal, 1,
                         &h_output);
            if (((signal_size*2-1)/2+i) < signal_size*2-1) return_data[(signal_size*2-1)/2+i] = h_output;
            return_data[(signal_size*2-1)/2-i] = h_output;
        }
    }
    else if  (strcmp(mode, "same") == 0) {

        int dims[1]={signal_size};
        return_object = (PyArrayObject *) PyArray_FromDims(1,dims,NPY_CDOUBLE);
        DoubleComplex *return_data  = (DoubleComplex *)PyArray_DATA(return_object);

        for (int i=0; i< signal_size/2+1; i++){
            hipblasZdotc(handle, signal_size-i,
                         &d_signal[i], 1,
                         d_signal, 1,
                         &h_output);

            if ((signal_size/2+i) < signal_size) return_data[signal_size/2+i] = h_output;
            return_data[signal_size/2-i] = h_output;
        }

    }
    else if  (strcmp(mode, "valid") == 0) {

        int dims[1]={1};
        return_object = (PyArrayObject *) PyArray_FromDims(1,dims,NPY_CDOUBLE);
        DoubleComplex *return_data  = (DoubleComplex *)PyArray_DATA(return_object);

        hipblasZdotc(handle, signal_size,
                     d_signal, 1,
                     d_signal, 1,
                     &h_output);
        return_data[0] = h_output;
    }
   else {
        PyErr_SetString(PyExc_TypeError, "this mode do not exist");
        PyErr_Print();
   }

    // Finish cublas
    hipblasDestroy(handle);
    hipFree(d_signal);
    Py_DECREF(h_signal_array);

    return(PyArray_Return(return_object));
}



static char extension_docs_sp[] =
    "autocorrelation(signal)\nAutocorrelation function implemented in CUDA\n(Single precision)\n  ";

static char extension_docs_dp[] =
    "autocorrelation(signal)\nAutocorrelation function implemented in CUDA\n(Double precision)\n  ";


static PyMethodDef extension_funcs[] =
{
    {"acorrelate", (PyCFunction) autocorrelation_sp, METH_VARARGS|METH_KEYWORDS, extension_docs_sp},
    {"dacorrelate", (PyCFunction) autocorrelation_dp, METH_VARARGS|METH_KEYWORDS, extension_docs_dp},
    {NULL}
};


PyMODINIT_FUNC initgpu_correlate(void)
{
//  Importing numpy array types
    import_array();
    Py_InitModule3("gpu_correlate", extension_funcs,
                   "Autocorrelation functions (CUDA)");
};